#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    int device;
    hipError_t error = hipGetDevice(&device);
    if (error != hipSuccess) {
        printf("hipMalloc failed: %s\n", hipGetErrorString(error));
        return 1;
    }

    struct hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device);

    printf("Device %d: %s\n", device, props.name);
    printf("Compute Capability: %d.%d\n", props.major, props.minor);

    return 0;
}
